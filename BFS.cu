#include "hip/hip_runtime.h"
/**
 * The MIT License (MIT)
 *
 * Copyright (c) 2015 Yichao Cheng
 * 
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 * 
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 * 
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 * THE SOFTWARE.
 */


/**
 * Test the bfs implementation
 *
 * Author: Yichao Cheng (onesuperclark@gmail.com)
 * Created on: 2014-10-28
 * Last Modified: 2014-12-18
 */

#include "oliver.h"

FILE * outputFile;

struct BFS_Vertex {
    int level;

    inline void print() {
        fprintf(outputFile, "%d\n", level);
    }
};

struct BFS_edge_F {
    __device__
    inline int gather(BFS_Vertex src, EdgeId outdegree, Dump_Edge edge) {
        return src.level + 1;        
    }

    __device__
    inline void reduce(int &accumulator, int accum) {
        accumulator = accum; // benign race happens
    }
};  // edgeMap

struct BFS_vertex_F {
    int infiniteCost;

    BFS_vertex_F(int _inf) : infiniteCost(_inf) {}

    __device__
    inline void update(BFS_Vertex &v, int accum) { v.level = accum; }

    __device__
    inline bool cond(BFS_Vertex v, int accum) {
        return (v.level == infiniteCost);
    }
};  // vertexFilter

struct BFS_init_F {
    int level;

    BFS_init_F(int _level) : level(_level) {}

    __device__
    inline void operator() (BFS_Vertex &v, int accum) { v.level = level; }
};  // vertexMap


int main(int argc, char **argv) {
    CommandLine cl(argc, argv, "<inFile> [-dimacs] [-verbose] [-round 100]");
    char * inFile = cl.getArgument(0);
    VertexId source = cl.getOptionIntValue("-s", 0);
    int max_rounds = cl.getOptionIntValue("-round", 100);
    bool dimacs = cl.getOption("-dimacs");
    bool verbose = cl.getOption("-verbose");
    int group_size = cl.getOptionIntValue("-g", 1);
    bool use_scan = cl.getOption("-scan");

    // Read the graph file.
    CsrGraph<int, int> graph;
    if (dimacs) {
        graph.fromDimacsFile(inFile);
    } else {
        graph.fromEdgeListFile(inFile);
    }

    // Algorithm specific parameters
    const int infCost = 0x7fffffff;

    Oliver<BFS_Vertex, Dump_Edge, int> ol;
    ol.readGraph(graph);

    // Initializes the value of all vertices.
    VertexSubset all(graph.vertexCount, true);
    ol.vertexMap<BFS_init_F>(all, BFS_init_F(infCost));
    all.del();  // No longer used

    // Make a dense VertexSubset with a singleton vertex (source)
    // and initializes the level of it to 0
    VertexSubset frontier(graph.vertexCount, source);
    ol.vertexMap<BFS_init_F>(frontier, BFS_init_F(0));

    // Sparse VertexSubset to represent the expanding edges.
    VertexSubset edgeFrontier(graph.vertexCount, false); 

    double start = getTimeMillis();    
    Stopwatch w;
    w.start();

    int iterations = 0;
    while (1) {
        int size = frontier.size();
        
        switch(group_size) {
            case 1:  ol.edgeFilter<BFS_edge_F, 1>(edgeFrontier, frontier, BFS_edge_F()); break;
            case 2:  ol.edgeFilter<BFS_edge_F, 2>(edgeFrontier, frontier, BFS_edge_F()); break;
            case 4:  ol.edgeFilter<BFS_edge_F, 4>(edgeFrontier, frontier, BFS_edge_F()); break;
            case 8:  ol.edgeFilter<BFS_edge_F, 8>(edgeFrontier, frontier, BFS_edge_F()); break;
            case 16: ol.edgeFilter<BFS_edge_F, 16>(edgeFrontier, frontier, BFS_edge_F()); break;
            case 32: ol.edgeFilter<BFS_edge_F, 32>(edgeFrontier, frontier, BFS_edge_F()); break;
            default: assert(0);
        }

        if (use_scan) 
            ol.vertexFilter<BFS_vertex_F, true>(frontier, edgeFrontier, BFS_vertex_F(infCost));
        else
            ol.vertexFilter<BFS_vertex_F, false>(frontier, edgeFrontier, BFS_vertex_F(infCost));
  
        if (size == 0 || iterations == max_rounds) break;
        if (verbose) {
            LOG(INFO) << "BFS iterations " << iterations <<", size: "<< size
                      <<", time: " << w.getElapsedMillis() << "ms";
        }
        iterations++;
    }

    double totalTime =  getTimeMillis() - start;
    LOG(INFO) << "iterations: "<< iterations <<", time: " << totalTime << "ms";

    // Log the vertex value into a file
    outputFile = fopen("BFS.txt", "w");
    ol.printVertices();

    frontier.del();
    edgeFrontier.del();
    return 0;
}
