/**
 * An interface for CPU/GPU hybrid memory management
 *
 * Author: Yichao Cheng (onesuperclark@gmail.com)
 * Created on: 2014-10-22
 * Last Modified: 2014-10-22
 *
 */

#include "olive_def.h"
#include "olive_mem.h"


error_t olive_malloc(void ** ptr, size_t size, olive_mem_t type) {
    switch (type) {
    case OLIVE_MEM_HOST:
        * ptr = malloc(size);
        if (* ptr == NULL) return FAILURE;
        break;
    case OLIVE_MEM_HOST_PINNED:
        if (hipHostMalloc(ptr, size, hipHostMallocPortable) != hipSuccess) return FAILURE;
        break;
    case OLIVE_MEM_HOST_MAPPED:
        unsigned int flags = hipHostMallocPortable;
        // Maps the allocation into the CUDA address space. The device pointer to 
        // the memory may be obtained by calling hipHostGetDevicePointer().
        flags |= hipHostMallocMapped;
        // WriteCombined memory can be transferred across the PCI Express bus more quickly 
        // on some system configurations, but cannot be read efficiently by most CPUs.
        // So it is a good option for host->device transfers.
        flags |= hipHostMallocWriteCombined;
        if (hipHostMalloc(ptr, size, flags) != hipSuccess) return FAILURE;
        break;
    case OLIVE_MEM_DEVICE:
        if (hipMalloc(ptr, size) != hipSuccess) return FAILURE;
        break;
    default:
        olive_fatal("invalid memory type");
    }
    return SUCCESS;
 }

error_t olive_calloc(void** ptr, size_t size, totem_mem_t type) {
    if (olive_malloc(ptr, size, type) != SUCCESS) return FAILURE;
    switch (type) {
    case TOTEM_MEM_HOST:
    case TOTEM_MEM_HOST_PINNED:
    case TOTEM_MEM_HOST_MAPPED:
        memset(* ptr, 0, size);
        break;
    case TOTEM_MEM_DEVICE:
        if (hipMemset(* ptr, 0, size) != hipSuccess) return FAILURE;
        break;
    default:
        olive_fatal("invalid memory type");
    }
    return SUCCESS;
}

void olive_free(void * ptr, olive_mem_t type) {
    switch (type) {
    case OLIVE_MEM_HOST:
        free(ptr);
        break;
    case OLIVE_MEM_HOST_PINNED:
    case OLIVE_MEM_HOST_MAPPED:
        CUT_SAFE_CALL(hipHostFree(ptr));
        break;
    case OLIVE_MEM_DEVICE:
        CUT_SAFE_CALL(hipFree(ptr));
        break;
    default:
        olive_fatal("invalid memory type");
    }
}

 
