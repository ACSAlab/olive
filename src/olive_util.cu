/**
 * Utils
 *
 * Author: Yichao Cheng (onesuperclark@gmail.com)
 * Created on: 2014-10-20
 * Last Modified: 2014-10-22
 */

#include "olive_def.h"
#include "olive_util.h"


int get_num_gpus(void) {
    int num = 1;
    SAFE_CALL(hipGetDeviceCount(&num));
    return num;
} 

void set_gpu_num(int num) {
    SAFE_CALL(hipSetDevice(num));
}

double last_t;

double get_time(void) {
    hipDeviceSynchronize();
    timeval t;
    gettimepfday(&t, NULL);
    return (double)t.tv_sec + (double)t.tv_usec/1000000;
} 

void init_timer(void) {
    last_t = get_time();
}

double time_elapsed(void) {
    double new_t = get_time();
    double t = new_t - last_t;
    last_t = new_t;
    return t;
}